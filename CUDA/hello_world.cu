
#include <hip/hip_runtime.h>
#include <stdio.h>


/*__global__ says the function is a kernel
will be executed on the GPU by one or more
simultaneous threads when called.*/

__global__ void myKernel(){
};

int main(){
 /*notes <<<nblocks,nthreadsperblock>>>*/
 myKernel<<<1, 1>>>();
 printf("Hello, World!\n");
 return 0;
}
