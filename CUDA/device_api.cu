#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define N 5

__global__ void setup_kernel(hiprandState *state){
       int id = threadIdx.x + blockIdx.x * N;
       
       /* Each thread gets same seed, a different sequence number, no
       offset */

       hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state, int *result){
       int id = threadIdx.x + blockIdx.x * N;
       int count = 0;
       unsigned int x;

       //Copy state to local memory for efficiency


       hiprandState localState = state[id];

       //Generate pseudo-random unsigned ints

       for (int n = 0; n < 100000; n++){
         x = hiprand(&localState);
         /* Check if odd */
         if (x & 1){
            count ++;
         }
       }

       //copy state back to global 
      state[id] = localState;

       //store results
 
       result[id] += count; 
}


int main(int argc, char *argv[]){
    int i, total;
    int *devResults, *hostResults;
    hiprandState *devStates;

    // Allocate space for results on host

    hostResults = (int *) calloc(N*N,sizeof(int));
   
   //Allocate space for results on device
 
   hipMalloc((void**) &devResults,N*N*sizeof(int));
  
   //set results to 0 in the device

   hipMemset(devResults,0,N*N*sizeof(int));
  
   //allocate space for prng states on device */

   hipMalloc((void **) &devStates, N * N * sizeof(hiprandState));

   //set-up prng states

   setup_kernel<<<N,N>>>(devStates);

   /* Generate and use pseudorandom numbers*/

   for(i=0; i < 10; i++){
     generate_kernel<<<N,N>>>(devStates, devResults);
   }

   // copy device memory to host

   hipMemcpy(hostResults, devResults, N*N*sizeof(int),hipMemcpyDeviceToHost);

   //show results
   total = 0;
   for(i=0; i < N*N; i++){
      total += hostResults[i];
   }
   printf("fraction odd was %10.13f\n", (float) total / (5.0 * 5.0 * 100000.0 * 10.0));

   //clean-up 

   hipFree(devStates);
   hipFree(devResults);
   free(hostResults);

   return 0;
}
