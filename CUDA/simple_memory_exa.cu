//pairwise-sum the elements of vector v and store the results in v[0]

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <unistd.h>
#include <time.h>


/* 
 *this program computes the sum of the elements of 
 *vector v using the pairwise (cascading) sum algorithm.
 */

#define N 8 //length of vector v... must be a power of 2

//fill the vector v with n random floating point numbers.

void vfill(float* v, int n){
  int i;
  for (i=0; i<n;i++){
   v[i] = (float) rand() / RAND_MAX;
  }
};

float lsum(float* v, int n){
  int i;
  float result = 0;
  for (i=0; i<n;i++){
   result  = result + v[i];
  }
  return result;
};

//print the vector v

void vprint(float*v , int n){
  int i;
  printf("v = \n");
  for (i=0 ; i < n; i++){
    printf("%7.3f\n",v[i]);
}
  printf("\n");
};

__global__ void psum(float* v){
  int t = threadIdx.x; // thread index
  int n = blockDim.x; // should be half the length of v.
                      // number of threads per block in x direction.
  while (n != 0){
    if (t < n){
       v[t] = v[t] + v[t+n];}
    __syncthreads(); 
    n = n/2;
  }
};

int main(void){
  float *v_h, *v_d; //host and device copies of our vector, respectively
  //dynamically allocate memory on the host for v_h

  v_h = (float*) malloc(N * sizeof(*v_h));
  
  // dynamically allocate memory on the device for v_d
  hipMalloc((float**)&v_d, N * sizeof(*v_d));

  //fill v_h with N RANDOM floating point numbers
  vfill(v_h,N);
  //print v_h to the console
  vprint(v_h,N);
  // write the contents of v_h to v_d
  hipMemcpy(v_d,v_h,N*sizeof(float),hipMemcpyHostToDevice);
  //time the linear approach on the cpu

  clock_t start = clock();
  float s = lsum(v_h, N);
  float elapsedTime = ((float) clock() -start) / CLOCKS_PER_SEC;
  printf("Linear Sum = %7.3f, CPU time elapsed: %f seconds\n", s, elapsedTime);
  //time GPU implementation of pairwise implementation.
 //compute the pairwise sum of the elements of v_d and store the result in v_d[0]
  float gpuElapsedTime;
  hipEvent_t gpuStart, gpuStop;
  hipEventCreate(&gpuStart);
  hipEventCreate(&gpuStop);
  hipEventRecord(gpuStart,0);
  psum <<<1,N/2>>>(v_d);
  hipEventRecord(gpuStop,0);
  hipEventSynchronize(gpuStop);
  hipEventElapsedTime(&gpuElapsedTime, gpuStart, gpuStop); //time is in milliseconds.
  hipEventDestroy(gpuStart);
  hipEventDestroy(gpuStop);
  //write the pairwise sum, v_d[0], to v_h[0]
  hipMemcpy(v_h,v_d,sizeof(float),hipMemcpyDeviceToHost);

  printf("pairwise sum  = %7.3f, GPU time elapsed time %f\n", v_h[0],gpuElapsedTime/1000);
  //free dynamically-allocated host memory
  free(v_h);
  //free dynamically-allocated device memory
  hipFree(v_d);
}
