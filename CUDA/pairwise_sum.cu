//pairwise-sum the elements of vector v and store the results in v[0]

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


/* 
 *this program computes the sum of the elements of 
 *vector v using the pairwise (cascading) sum algorithm.
 */

#define N 8 //length of vector v... must be a power of 2

//fill the vector v with n random floating point numbers.

void vfill(float* v, int n){
  int i;
  for (i=0; i<n;i++){
   v[i] = (float) rand() / RAND_MAX;
  }
};

//print the vector v

void vprint(float*v , int n){
  int i;
  printf("v = \n");
  for (i=0 ; i < n; i++){
    printf("%7.3f\n",v[i]);
}
  printf("\n");
};

__global__ void psum(float* v){
  int t = threadIdx.x; // thread index
  int n = blockDim.x; // should be half the length of v.
                      // number of threads per block in x direction.
  while (n != 0){
    if (t < n){
       v[t] = v[t] + v[t+n];}
    __syncthreads(); 
    n = n/2;
  }
};

int main(void){
  float *v_h, *v_d; //host and device copies of our vector, respectively
  //dynamically allocate memory on the host for v_h

  v_h = (float*) malloc(N * sizeof(*v_h));
  
  // dynamically allocate memory on the device for v_d
  hipMalloc((float**)&v_d, N * sizeof(*v_d));

  //fill v_h with N RANDOM floating point numbers
  vfill(v_h,N);
  //print v_h to the console
  vprint(v_h,N);
  // write the contents of v_h to v_d
  hipMemcpy(v_d,v_h,N*sizeof(float),hipMemcpyHostToDevice);
 //compute the pairwise sum of the elements of v_d and store the result in v_d[0]
 psum <<<1,N/2>>>(v_d);
 //write the pairwise sum, v_d[0], to v_h[0]
 hipMemcpy(v_h,v_d,sizeof(float),hipMemcpyDeviceToHost);

 printf("pairwise sum= %7.3f\n",v_h[0]);
 //free dynamically-allocated host memory
 free(v_h);
 //free dynamically-allocated device memory
 hipFree(v_d);
}
