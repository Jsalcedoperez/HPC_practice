/* this program uses the host CURAND API to generate
10 pseudorandom floats*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

int main(int argc, char *argv[]){

  size_t n = 10;
  size_t i;
  hiprandGenerator_t gen;
  float *devData, *hostData;
  
  /* Allocate n floats on host */
  //hostData = (float*) calloc(n,sizeof(float));

   hostData = (float*) malloc(n*sizeof(*hostData));
 
  //Allocate n floats on device

  hipMalloc((void**) &devData, n*sizeof(*devData));
  
  //create a Memsenne Twister pseudorandom number generator

  hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_MTGP32);
  
  //set seed

  hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

  //Generate n floats on device

  hiprandGenerateUniform(gen, devData, n);

  //copy device memory to host

  hipMemcpy(hostData, devData, n * sizeof(float),hipMemcpyDeviceToHost);

 //show results

  printf("Random Uni(0,1) draws:\n");

   for (i=0;i<n;i++){
     printf(" %1.4f\n", hostData[i]);
   }

   printf("\n");

   /* Cleanup */

   hiprandDestroyGenerator(gen);
   hipFree(devData);
   free(hostData);

   //return 0;
}
