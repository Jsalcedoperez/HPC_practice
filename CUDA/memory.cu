#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define N 16
#define threadsPerBlock 4
#define blocksPerGrid 2

//fill the vector v with n random floating point numbers.

void vfill(float* v, int n){
  int i;
  for (i=0; i<n;i++){
   v[i] = (float) rand() / RAND_MAX;
  }
};

//print the vector v

void vprint(float*v , int n){
  int i;
  printf("v = \n");
  for (i=0 ; i < n; i++){
    printf("%7.3f\n",v[i]);
}
  printf("\n");
};


__global__ void dot(float* a, float* b, float* partial_c){

__shared__ float cache[threadsPerBlock];

int tid = threadIdx.x + blockIdx.x * blockDim.x;
int cacheIndex = threadIdx.x;
float temp = 0;

while (tid < N){
  temp += a[tid] + b[tid];
  tid += blockDim.x * gridDim.x;
}

cache[cacheIndex] = temp;

//synchronize
__syncthreads();

int i = blockDim.x/2;

while (i != 0) {

  if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex + 1];}
  __syncthreads();
  i = i/2;
}

//record the result in partial_c

if (cacheIndex == 0){
  partial_c[blockIdx.x] = cache[0];
}
};

int main(void){
  float c;
  float *a, *b, *partial_c;
  float *a_d, *b_d, *partial_c_d;
  
  a = (float*) malloc(N*sizeof(*a));
  b = (float*) malloc(N*sizeof(*b));
  partial_c = (float*) malloc(blocksPerGrid*sizeof(*partial_c));


  hipMalloc((float**) &a_d, (N*sizeof(a_d)));
  hipMalloc((float**) &b_d, (N*sizeof(b_d)));
  hipMalloc((float**) &partial_c_d, (blocksPerGrid*sizeof(partial_c_d)));

  //fill v_h with N RANDOM floating point numbers
  vfill(a,N);
  vfill(b,N);
  //print v_h to the console
  vprint(a,N);
  vprint(b,N);

  hipMemcpy(a_d,a,threadsPerBlock*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(b_d,b,threadsPerBlock*sizeof(float),hipMemcpyHostToDevice);

  dot<<<blocksPerGrid,threadsPerBlock>>>(a_d,b_d,partial_c_d);

  hipMemcpy(partial_c,partial_c_d,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);
  c = 0;

  for (int i = 0; i<blocksPerGrid; i++){
      c += partial_c[i];
  }

printf("this is c: %.3f\n",c);
return 0;
}
